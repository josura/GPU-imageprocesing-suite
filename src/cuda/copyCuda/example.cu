#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <stdarg.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>


using namespace std;


/* Check a CUDA error status, printing a message and exiting
 * in case of failure
 */
#define BUFSIZE 4096
void cuda_check(hipError_t err, const char *msg, ...) {
        if (err != hipSuccess) {
                char msg_buf[BUFSIZE + 1];
                va_list ap;
                va_start(ap, msg);
                vsnprintf(msg_buf, BUFSIZE, msg, ap);
                va_end(ap);
                msg_buf[BUFSIZE] = '\0';
                fprintf(stderr, "%s - error %d (%s)\n", msg_buf, err, hipGetErrorString(err));
                exit(1);
        }
}


static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
        if(err!=hipSuccess)
        {
                fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
                std::cin.get();
                exit(EXIT_FAILURE);
        }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)




texture<uchar4, 2, hipReadModeNormalizedFloat> tex;


__global__
void imgcopy(uchar4 * input,uchar4 *output, int width, int height, int output_pitch_el)
{
        int row = blockDim.y*blockIdx.y + threadIdx.y;
        int col = blockDim.x*blockIdx.x + threadIdx.x;

        if (row < height && col < width) {
                float4 px = tex2D(tex, col, row);
                output[row*output_pitch_el+col] =
                        make_uchar4(px.x*255, px.y*255, px.z*255, px.w*255);
        }
}


__global__
void imgCopy(unsigned char* input, 
                                        unsigned char* output, 
                                        int width,
                                        int height,
                                        int colorWidthStep){
	//2D Index of current thread
        const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
        const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

        //Only valid threads perform memory I/O
        if((xIndex<width) && (yIndex<height))
        {
                //Location of colored pixel in input
                const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
                

                output[color_tid]           = input[color_tid];
                output[color_tid+1]           = input[color_tid + 1];
                output[color_tid+2]           = input[color_tid + 2];

        }
}


void imageCopy(const cv::Mat& input, cv::Mat& output)
{
        //Calculate total number of bytes of input and output image
        const int colorBytes = input.step * input.rows;

        unsigned char *d_input, *d_output;

        //Allocate device memory
        SAFE_CALL(hipMalloc<unsigned char>(&d_input,colorBytes),"CUDA Malloc Failed");
        SAFE_CALL(hipMalloc<unsigned char>(&d_output,colorBytes),"CUDA Malloc Failed");

        //Copy data from OpenCV input image to device memory
        SAFE_CALL(hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");

        //Specify a reasonable block size
        const dim3 block(16,16);

        //Calculate grid size to cover the whole image
        const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

        //Launch the color conversion kernel
        imgCopy<<<grid,block>>>(d_input,d_output,input.cols,input.rows,input.step);

        //Synchronize to check for any kernel launch errors
        SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

        //Copy back data from destination device meory to OpenCV output image
        SAFE_CALL(hipMemcpy(output.ptr(),d_output,colorBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

        //Free the device memory
        SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
        SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}




int main(int argc, char ** argv){
	std::string imagePath = "../img/gallo.png";

        //Read input image from the disk
        cv::Mat input = cv::imread(imagePath,cv::IMREAD_COLOR);

        if(input.empty())
        {
                std::cerr<<"Image Not Found!"<<std::endl;
                return -1;
        }

        //Create output image
        //std::cout<<cv::CV_8UC1<<std::endl;
        cv::Mat output(input.rows,input.cols,input.type());
        if (output.empty())
        {
                cout << "\n Image not created. You"
                     " have done something wrong. \n";
                return -1;    // Unsuccessful.
        }


        //Call the wrapper function
        imageCopy(input,output);

        cv::imwrite("starry_night.png",output);


}
